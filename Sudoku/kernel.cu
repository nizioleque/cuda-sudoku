#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <fstream>
#include <string>
#include <iostream>
#include "utils.cuh"
#include "cpu.cuh"
#include "gpu.cuh"

const char* boardFilename = "sudoku.txt";

bool solveBoard(char* board, int index);

int main()
{
	std::ifstream inFS;
	inFS.open(boardFilename);
	clock_t cpuStart, cpuEnd, gpuStart, gpuEnd;


	std::string line;
	getline(inFS, line);
	int nBoards = stoi(line);

	char* boards = new char[nBoards * 9 * 9];
	int boardsArrayIndex = 0;

	for (int board = 0; board < nBoards; board++) {
		for (int row = 0; row < 9;) {
			getline(inFS, line);
			if (line.length() == 0) continue;
			for (int column = 0; column < 9; column++) {
				boards[boardsArrayIndex++] = line[column] - '0';
			}
			row++;
		}
	}

	cpuStart = clock();
	solveCpu(boards, nBoards);
	cpuEnd = clock();

	gpuStart = clock();
	int result = solveGpu(boards, nBoards);
	gpuEnd = clock();

	printf("CPU time: %f\nGPU time: %f\n", ((double)cpuEnd - cpuStart) / CLOCKS_PER_SEC, ((double)gpuEnd - cpuEnd) / CLOCKS_PER_SEC);

	delete[] boards;

	return result;

}


